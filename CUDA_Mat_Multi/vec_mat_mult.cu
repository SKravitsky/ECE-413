#include "hip/hip_runtime.h"
/* Vector-matrix multiplication: Y = A * X.
 * Host code.
 * Author: Naga Kandasamy
 * Date: 2/21/2017
*/

#include <stdlib.h>
#include <stdio.h>
#include <time.h>
#include <string.h>
#include <math.h>


#include "vec_mat_mult_kernel.cu"

#define MIN_NUMBER 1
#define MAX_NUMBER 4


extern "C" void compute_gold(float*, const float*, const float*, unsigned int, unsigned int);
Matrix allocate_matrix_on_gpu(const Matrix);
Matrix allocate_matrix(int, int, int);
void copy_matrix_to_device(Matrix, const Matrix);
void copy_matrix_from_device(Matrix, const Matrix);
void vec_mat_mult_on_device_using_global_memory(const Matrix, const Matrix, Matrix);
void vec_mat_mult_on_device_using_shared_memory(const Matrix, const Matrix, Matrix);
void print_matrix(const Matrix);
float get_random_number(int, int);
int checkResults(float *, float *, int, float);


int 
main(int argc, char** argv) {
	// Matrices for the program
	Matrix  A; // N x N matrix
	Matrix  X; // N x 1 vector
	Matrix  Y_cpu, Y_gpu_1, Y_gpu_2; // N x 1 vector
	
	// Initialize the random number generator with a seed value 
	srand(time(NULL));
	
	// Check command line arguments
	if(argc > 1){
		printf("Error. This program accepts no arguments. \n");
		exit(0);
	}		
	 
	// Allocate and initialize the matrices
	A  = allocate_matrix(MATRIX_SIZE, MATRIX_SIZE, 1); // Create a random N x N matrix
	X  = allocate_matrix(MATRIX_SIZE, 1, 1); // Create a random N x 1 vector 
	Y_cpu  = allocate_matrix(MATRIX_SIZE, 1, 0); // Allocate memory for the output vectors
	Y_gpu_1 = allocate_matrix(MATRIX_SIZE, 1, 0); 
    Y_gpu_2 = allocate_matrix(MATRIX_SIZE, 1, 0);
 
    // compute the vector-matrix multiplication on the CPU for comparison    	
	compute_gold(Y_cpu.elements, A.elements, X.elements, A.num_rows, A.num_columns);
	
	// Perform the vector-matrix multiplication on the GPU using global memory
    // Return the results in Y_gpu_1
	vec_mat_mult_on_device_using_global_memory(A, X, Y_gpu_1);
   
	// check if the device result is equivalent to the expected solution
    printf("Checking against reference result. \n");
	int size_elements = NUM_ROWS;
	int res = checkResults(Y_cpu.elements, Y_gpu_1.elements, size_elements, 0.0001);
	printf("Test %s\n", (1 == res) ? "PASSED" : "FAILED");


    // Perform the vector-matrix multiplication on the GPU using shared memory
    // Return the results in Y_gpu_2
	vec_mat_mult_on_device_using_shared_memory(A, X, Y_gpu_2);
   
	// check if the device result is equivalent to the expected solution
    printf("Checking against reference result. \n");
    res = checkResults(Y_cpu.elements, Y_gpu_2.elements, size_elements, 0.0001);
	printf("Test %s\n", (1 == res) ? "PASSED" : "FAILED");

	// Free host matrices
	free(A.elements); A.elements = NULL;
	free(X.elements); X.elements = NULL;
	free(Y_cpu.elements); Y_cpu.elements = NULL;
	free(Y_gpu_1.elements); Y_gpu_1.elements = NULL;
    free(Y_gpu_2.elements); Y_gpu_2.elements = NULL;

	return 0;
}

// Complete the functionality of vector-matrix multiplication using the GPU 
// Kernel should use global memory
void 
vec_mat_mult_on_device_using_global_memory(const Matrix A, const Matrix X, Matrix Y)
{

}

// Complete the functionality of vector-matrix multiplication using the GPU
// Kernel should use shared memory
void 
vec_mat_mult_on_device_using_shared_memory(const Matrix A, const Matrix X, Matrix Y)
{

}


// Allocate a device matrix of same size as M.
Matrix 
allocate_matrix_on_gpu(const Matrix M)
{
    Matrix Mdevice = M;
    int size = M.num_rows * M.num_columns * sizeof(float);
    hipMalloc((void**)&Mdevice.elements, size);
    return Mdevice;
}

// Allocate a matrix of dimensions height*width
//	If init == 0, initialize to all zeroes.  
//	If init == 1, perform random initialization.
Matrix 
allocate_matrix(int num_rows, int num_columns, int init)
{
    	Matrix M;
    	M.num_columns = M.pitch = num_columns;
    	M.num_rows = num_rows;
    	int size = M.num_rows * M.num_columns;
		
	M.elements = (float*) malloc(size*sizeof(float));
	for(unsigned int i = 0; i < size; i++){
		if(init == 0) M.elements[i] = 0; 
		else
			M.elements[i] = get_random_number(MIN_NUMBER, MAX_NUMBER);
	}
    return M;
}	

// Copy a host matrix to a device matrix.
void 
copy_matrix_to_device(Matrix Mdevice, const Matrix Mhost)
{
    int size = Mhost.num_rows * Mhost.num_columns * sizeof(float);
    Mdevice.num_rows = Mhost.num_rows;
    Mdevice.num_columns = Mhost.num_columns;
    Mdevice.pitch = Mhost.pitch;
    hipMemcpy(Mdevice.elements, Mhost.elements, size, hipMemcpyHostToDevice);
}

// Copy a device matrix to a host matrix.
void 
copy_matrix_from_device(Matrix Mhost, const Matrix Mdevice)
{
    int size = Mdevice.num_rows * Mdevice.num_columns * sizeof(float);
    hipMemcpy(Mhost.elements, Mdevice.elements, size, hipMemcpyDeviceToHost);
}

// Prints the matrix out to screen
void 
print_matrix(const Matrix M)
{
	for(unsigned int i = 0; i < M.num_rows; i++){
		for(unsigned int j = 0; j < M.num_columns; j++)
			printf("%f ", M.elements[i*M.num_columns + j]);
		printf("\n");
	} 
	printf("\n");
}

// Returns a random floating-point number between the specified min and max values 
float 
get_random_number(int min, int max){
	return (float)floor((double)(min + (max - min + 1)*((float)rand()/(float)RAND_MAX)));
}

int 
checkResults(float *reference, float *gpu_result, int num_elements, float threshold)
{
    int checkMark = 1;
    float epsilon = 0.0;
    
    for(int i = 0; i < num_elements; i++)
        if(fabsf((reference[i] - gpu_result[i])/reference[i]) > threshold){
            checkMark = 0;
            break;
        }

    for(int i = 0; i < num_elements; i++)
        if(fabsf((reference[i] - gpu_result[i])/reference[i]) > epsilon){
            epsilon = fabsf((reference[i] - gpu_result[i])/reference[i]);
        }

    printf("Max epsilon = %f. \n", epsilon); 
    return checkMark;
}


