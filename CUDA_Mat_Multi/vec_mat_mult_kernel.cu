#include "hip/hip_runtime.h"
/* Vector-Matrix multiplication: Y = A * X.
 * Device code.
 */

#ifndef _MATRIXMUL_KERNEL_H_
#define _MATRIXMUL_KERNEL_H_

#include "vec_mat_mult.h"

/* Write the kernel for vector-matrix multiplication using GPU global memory. */
__global__ void vec_mat_kernel_naive(float *Ad, float *Xd, float *Yd)
{
	int tid = threadIdx.x + blockIdx.x*blockDim.x;
	float final = 0;

	for(unsigned int i = 0; i < MATRIX_SIZE; i++)
	{
		float a = Ad[tid * MATRIX_SIZE + i];
		float b = Xd[i];
		final += a * b;
	}

	Yd[tid] = final;

}


/* Write the kernel for vector-matrix multiplication using GPU shared memory. */
__global__ void vec_mat_kernel_optimized(float *Ad, float *Xd, float *Yd)
{
	__shared__ float a[16][16];
	__shared__ float b[16];

	int tx = threadIdx.x;
	int ty = threadIdx.y;
	int row = blockIdx.y*blockDim.y;

	int final = 0;

	for(unsigned int i = 0; i < MATRIX_SIZE; i += 16)
	{
		a[ty][tx] = Ad[row * MATRIX_SIZE + i + tx];
		b[tx] = Xd[tx + i];

		__syncthreads();
		if(threadIdx.x==0)
		{
			for(unsigned int j =0; j < blockDim.x; j++)
			{
				final += a[tx][j] * b[j];
			}
		}
		__syncthreads();
	}
	
	if(threadIdx.x == 0)
	{
		Yd[row] = final;
	}
	
}



#endif // #ifndef _MATRIXMUL_KERNEL_H_
