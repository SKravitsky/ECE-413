#include "hip/hip_runtime.h"
/* Vector-Matrix multiplication: Y = A * X.
 * Device code.
 */

#ifndef _MATRIXMUL_KERNEL_H_
#define _MATRIXMUL_KERNEL_H_

#include "vec_mat_mult.h"

/* Write the kernel for vector-matrix multiplication using GPU global memory. */
__global__ void vec_mat_kernel_naive(float *Ad, float *Xd, float *Yd)
{
	//Multiply A and X
}


/* Write the kernel for vector-matrix multiplication using GPU shared memory. */
__global__ void vec_mat_kernel_optimized(float *Ad, float *Xd, float *Yd)
{
	//Multiply A and X
}



#endif // #ifndef _MATRIXMUL_KERNEL_H_
