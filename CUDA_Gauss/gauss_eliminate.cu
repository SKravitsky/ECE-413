#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <sys/time.h>
#include <string.h>
#include <math.h>

// includes, kernels
#include "gauss_eliminate_kernel.cu"

#define BLOCK_SIZE 16
#define MIN_NUMBER 2
#define MAX_NUMBER 50

extern "C" int compute_gold(float*, const float*, unsigned int);
Matrix allocate_matrix_on_gpu(const Matrix M);
Matrix allocate_matrix(int num_rows, int num_columns, int init);
void copy_matrix_to_device(Matrix Mdevice, const Matrix Mhost);
void copy_matrix_from_device(Matrix Mhost, const Matrix Mdevice);
void gauss_eliminate_on_device(const Matrix M, Matrix P);
int perform_simple_check(const Matrix M);
void print_matrix(const Matrix M);
void write_matrix_to_file(const Matrix M);
float get_random_number(int, int);
void checkCUDAError(const char *msg);
int checkResults(float *reference, float *gpu_result, int num_elements, float threshold);


int 
main(int argc, char** argv) 
{
    // Matrices for the program
	Matrix  A; // The NxN input matrix
	Matrix  U; // The upper triangular matrix 
	
	// Initialize the random number generator with a seed value 
	srand(time(NULL));
	
	// Check command line arguments
	if(argc > 1){
		printf("Error. This program accepts no arguments. \n");
		exit(0);
	}		
	 
	// Allocate and initialize the matrices
	A  = allocate_matrix(MATRIX_SIZE, MATRIX_SIZE, 1);
	U  = allocate_matrix(MATRIX_SIZE, MATRIX_SIZE, 0); 

	// Perform Gaussian elimination on the CPU 
	Matrix reference = allocate_matrix(MATRIX_SIZE, MATRIX_SIZE, 0);

	int status = compute_gold(reference.elements, A.elements, A.num_rows);
	
	if(status == 0){
		printf("Failed to convert given matrix to upper triangular. Try again. Exiting. \n");
		exit(0);
	}
	status = perform_simple_check(reference); // Check that the principal diagonal elements are 1 
	if(status == 0){
		printf("The upper triangular matrix is incorrect. Exiting. \n");
		exit(0); 
	}
	printf("Gaussian elimination on the CPU was successful. \n");

	// Perform the vector-matrix multiplication on the GPU. Return the result in U
	gauss_eliminate_on_device(A, U);
    
	// check if the device result is equivalent to the expected solution
	int num_elements = MATRIX_SIZE*MATRIX_SIZE;
    int res = checkResults(reference.elements, U.elements, num_elements, 0.001f);
    printf("Test %s\n", (1 == res) ? "PASSED" : "FAILED");

	// Free host matrices
	free(A.elements); A.elements = NULL;
	free(U.elements); U.elements = NULL;
	free(reference.elements); reference.elements = NULL;

	return 0;
}


void 
gauss_eliminate_on_device(const Matrix A, Matrix U)
{
    struct timeval start, stop;

	Matrix A_device = allocate_matrix_on_gpu(A);
	hipMemset(A_device.elements, 0, A_device.num_rows * A_device.num_columns * sizeof(float));
	copy_matrix_to_device(A_device, A);

	int num_rows = ceil((float)A_device.num_rows / BLOCK_SIZE) / 8;

	dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE, 1);
	dim3 dimGrid(1, num_rows);

    gettimeofday(&start, NULL);
    hipDeviceSynchronize();
    for(unsigned int i = 0; i < A_device.num_rows - 1; i++)
    {
    	gauss_reduce_kernel <<< dimGrid, dimBlock >>> (A_device.elements, i, A_device.num_columns);
    	hipDeviceSynchronize();
    	gauss_eliminate_kernel <<< dimGrid, dimBlock >>> (A_device.elements, i, A_device.num_columns);
    	hipDeviceSynchronize();
    }
    gettimeofday(&stop, NULL);

    printf("GPU Execution time = %fus. \n", (float)(stop.tv_usec - start.tv_usec + (stop.tv_usec - start.tv_usec)/(float)1000000));

    copy_matrix_from_device(U, A_device);

    U.elements[(U.num_rows * U.num_columns) - 1] = 1;

    hipFree(A_device.elements);


}

// Allocate a device matrix of same size as M.
Matrix 
allocate_matrix_on_gpu(const Matrix M){
    Matrix Mdevice = M;
    int size = M.num_rows * M.num_columns * sizeof(float);
    hipMalloc((void**)&Mdevice.elements, size);
    return Mdevice;
}

// Allocate a matrix of dimensions height*width
//	If init == 0, initialize to all zeroes.  
//	If init == 1, perform random initialization.
Matrix 
allocate_matrix(int num_rows, int num_columns, int init){
    	Matrix M;
    	M.num_columns = M.pitch = num_columns;
    	M.num_rows = num_rows;
    	int size = M.num_rows * M.num_columns;
		
	M.elements = (float*) malloc(size*sizeof(float));
	for(unsigned int i = 0; i < size; i++){
		if(init == 0) M.elements[i] = 0; 
		else
            M.elements[i] = get_random_number(MIN_NUMBER, MAX_NUMBER);
	}
    return M;
}	

// Copy a host matrix to a device matrix.
void 
copy_matrix_to_device(Matrix Mdevice, const Matrix Mhost)
{
    int size = Mhost.num_rows * Mhost.num_columns * sizeof(float);
    Mdevice.num_rows = Mhost.num_rows;
    Mdevice.num_columns = Mhost.num_columns;
    Mdevice.pitch = Mhost.pitch;
    hipMemcpy(Mdevice.elements, Mhost.elements, size, hipMemcpyHostToDevice);
}

// Copy a device matrix to a host matrix.
void 
copy_matrix_from_device(Matrix Mhost, const Matrix Mdevice){
    int size = Mdevice.num_rows * Mdevice.num_columns * sizeof(float);
    hipMemcpy(Mhost.elements, Mdevice.elements, size, hipMemcpyDeviceToHost);
}

// Prints the matrix out to screen
void 
print_matrix(const Matrix M){
	for(unsigned int i = 0; i < M.num_rows; i++){
		for(unsigned int j = 0; j < M.num_columns; j++)
			printf("%f ", M.elements[i*M.num_rows + j]);
		printf("\n");
	} 
	printf("\n");
}

// Returns a random floating-point number between the specified min and max values 
float 
get_random_number(int min, int max){
	return (float)floor((double)(min + (max - min + 1)*((float)rand()/(float)RAND_MAX)));
}

// Performs a simple check on the upper triangular matrix. Checks to see if the principal diagonal elements are 1
int 
perform_simple_check(const Matrix M){
	for(unsigned int i = 0; i < M.num_rows; i++)
        if((fabs(M.elements[M.num_rows*i + i] - 1.0)) > 0.001) return 0;
	
    return 1;
} 

// Writes the matrix to a file 
void 
write_matrix_to_file(const Matrix M){
	FILE *fp;
	fp = fopen("matrix.txt", "wt");
	for(unsigned int i = 0; i < M.num_rows; i++){
        for(unsigned int j = 0; j < M.num_columns; j++)
            fprintf(fp, "%f", M.elements[i*M.num_rows + j]);
        }
    fclose(fp);
}

void 
checkCUDAError(const char *msg)
{
	hipError_t err = hipGetLastError();
	if( hipSuccess != err) 
	{
		printf("CUDA ERROR: %s (%s).\n", msg, hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}						 
}

int 
checkResults(float *reference, float *gpu_result, int num_elements, float threshold)
{
    int checkMark = 1;
    float epsilon = 0.0;
    
    for(int i = 0; i < num_elements; i++)
        if(fabsf((reference[i] - gpu_result[i])/reference[i]) > threshold){
            checkMark = 0;
            break;
        }

    for(int i = 0; i < num_elements; i++)
        if(fabsf((reference[i] - gpu_result[i])/reference[i]) > epsilon){
            epsilon = fabsf((reference[i] - gpu_result[i])/reference[i]);
        }

    printf("Max epsilon = %f. \n", epsilon); 
    return checkMark;
}
